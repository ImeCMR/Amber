#include "hip/hip_runtime.h"
//
// CUDA Linear Solvers with the CUSPARSE Library
// Only CSR matrices are supported.
//
// Aaron Luo & Ruxi Qi @ UC Irvine, Jul 2018
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <algorithm>
#include <sys/time.h> // For timing
#if defined(AMBER_PLATFORM_AMD)
#  include <hipblas.h>
#  include <hipsparse.h>
#  include "hip_definitions.h"
#else
#  include <hip/hip_runtime.h> // CUDA Runtime
#  include <hipsparse.h> // Using updated (v2) interfaces for CUBLAS and CUSPARSE
#  include <hipblas.h>
#endif
#include "hip/hip_runtime_api.h" // For error handling and device pickup

//
// Standard conjugate gradient method without preconditioner
// Matrix in the CSR format. Coded with the CUSPARSE library
// Both PBC and Non-PBC can be handled as long as the matrix
// is correctly set up by the CSR routine
//
extern "C" void cusparse_cg_wrapper_(float *x, float *b, int *I, int *J, float *val, int *xmymzm, int *nz_num, int *maxitn, float *accept, int *itn, float *residual)
{
    const int maxiter = *maxitn;
    const float tol = *accept; // accept is 1-norm in pbsa

    // input CSR matrix parameters
    int N = *xmymzm;
    int nz = *nz_num;

    // Create CUBLAS context
    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t = hipblasCreate(&cublasHandle);
    cublasErrorCheck(hipblasStatus_t);

    // Create CUSPARSE context 
    hipsparseHandle_t cusparseHandle = 0;
    hipsparseStatus_t cusparseStatus = hipsparseCreate(&cusparseHandle);
    cusparseErrorCheck(cusparseStatus);

    // Set up required arrays on device
    int *d_col, *d_row;
    float *d_val;
    float *d_x;
    float *d_r, *d_p, *d_q;
    cudaErrorCheck(hipMalloc((void **)&d_col, nz*sizeof(int)));
    cudaErrorCheck(hipMalloc((void **)&d_row, (N+1)*sizeof(int)));
    cudaErrorCheck(hipMalloc((void **)&d_val, nz*sizeof(float)));
    cudaErrorCheck(hipMalloc((void **)&d_x, N*sizeof(float)));
    cudaErrorCheck(hipMalloc((void **)&d_r, N*sizeof(float)));
    cudaErrorCheck(hipMalloc((void **)&d_p, N*sizeof(float)));
    cudaErrorCheck(hipMalloc((void **)&d_q, N*sizeof(float)));

    // Get data into working device arrays
    hipMemcpy(d_col, J, nz*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_row, I, (N+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_val, val, nz*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_r, b, N*sizeof(float), hipMemcpyHostToDevice);

    const float constONE = 1.0;
    const float constZERO = 0.0;    
#if __CUDACC_VER_MAJOR__ >= 11
    hipsparseSpMatDescr_t descr_A;
    hipsparseDnVecDescr_t descr_p, descr_q;
    void *spmvBuffer = NULL;
    size_t spmvBufferSize = 0;
    hipsparseCreateCsr(&descr_A, N, N, nz, d_row, d_col, d_val,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    hipsparseCreateDnVec(&descr_p, N, d_p, HIP_R_32F);
    hipsparseCreateDnVec(&descr_q, N, d_q, HIP_R_32F);
    hipsparseSpMV_bufferSize(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            &constONE, descr_A, descr_p, &constZERO, descr_q,
                            HIP_R_32F, 
#if __CUDACC_VER_MAJOR__ == 11 && __CUDACC_VER_MINOR__ <= 2
                            HIPSPARSE_MV_ALG_DEFAULT,
#else
                            HIPSPARSE_SPMV_ALG_DEFAULT,
#endif
                            &spmvBufferSize);
    hipMalloc(&spmvBuffer, spmvBufferSize);
#else    
    // Description of the A matrix as in Ax = b    
    hipsparseMatDescr_t descrA = 0;
    cusparseStatus = hipsparseCreateMatDescr(&descrA);
    cusparseErrorCheck(cusparseStatus);
    // Define the properties of the A matrix
    hipsparseSetMatType(descrA,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA,HIPSPARSE_INDEX_BASE_ZERO);
#endif
    
    // CG iteration
    float r0, r1; // initial 2-norm
    hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r0);
    r1 = r0;
    float b0; // initial 1-norm
    hipblasSasum(cublasHandle, N, d_r, 1, &b0);
    float alpha, beta; // CG initial parameters
    float dot, nalpha;
    int k;
    for (k = 1; k < maxiter; k++) { // iteration starts
        // compute beta & p
	beta = r1/r0;
	hipblasSscal(cublasHandle, N, &beta, d_p, 1);
	hipblasSaxpy(cublasHandle, N, &constONE, d_r, 1, d_p, 1) ;
        // compute Ap & alpha
#if __CUDACC_VER_MAJOR__ >= 11
        hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                     &constONE, descr_A, descr_p, &constZERO, descr_q,
                     HIP_R_32F, 
#if __CUDACC_VER_MAJOR__ == 11 && __CUDACC_VER_MINOR__ <= 2
                     HIPSPARSE_MV_ALG_DEFAULT, 
#else
                     HIPSPARSE_SPMV_ALG_DEFAULT,
#endif
                     spmvBuffer);
#else        
	hipsparseScsrmv(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &constONE, descrA, d_val, d_row, d_col, d_p, &constZERO, d_q);
#endif        
	hipblasSdot(cublasHandle, N, d_p, 1, d_q, 1, &dot);
	alpha = r1/dot;
        // update x
	hipblasSaxpy(cublasHandle, N, &alpha, d_p, 1, d_x, 1);
	nalpha = -alpha;
        // update r
	hipblasSaxpy(cublasHandle, N, &nalpha, d_q, 1, d_r, 1);
        // update norm and check convergence
	r0 = r1;
	hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
        //printf("itn %d residue %e tol %e init norm %e\n", k, tol, b0);
        if (r1 < tol*b0){
            break;
        }
    }

    hipMemcpy(x, d_x, N*sizeof(float), hipMemcpyDeviceToHost);
    *itn = k;
    *residual = r1;

#if __CUDACC_VER_MAJOR__ >= 11
    hipsparseDestroySpMat(descr_A);
    hipsparseDestroyDnVec(descr_p);
    hipsparseDestroyDnVec(descr_q);
    hipFree(spmvBuffer);
#endif    
    
    // Destroy contexts
    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);

    // Free device memory
    hipFree(d_col);
    hipFree(d_row);
    hipFree(d_val);
    hipFree(d_x);
    hipFree(d_r);
    hipFree(d_p);
    hipFree(d_q);

    // clean up all state, flush all profile data
    hipDeviceReset();

}

//
// Bi-conjugate gradient (BICGSTAB) method without preconditioner
// Matrix in the CSR format. Coded with the CUSPARSE library
// Both PBC and Non-PBC can be handled as long as the matrix
// is correctly set up by the CSR routine
//
extern "C" void cusparse_bicg_wrapper_(float *x, float *b, int *I, int *J, float *val, int *xmymzm, int *nz_num, int *maxitn, float *accept, int *itn, float *residual)
{
    const int maxiter = *maxitn;
    const float tol2 = *accept * *accept; // accept is 1-norm in pbsa

    // input CSR matrix parameters
    int N = *xmymzm;
    int nz = *nz_num;

    // Create CUBLAS context
    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t = hipblasCreate(&cublasHandle);
    cublasErrorCheck(hipblasStatus_t);

    // Create CUSPARSE context 
    hipsparseHandle_t cusparseHandle = 0;
    hipsparseStatus_t cusparseStatus = hipsparseCreate(&cusparseHandle);
    cusparseErrorCheck(cusparseStatus);

    // Set up required arrays on device
    int *d_col, *d_row;
    float *d_val;
    float *d_x, *d_p, *d_q;
    float *d_r, *d_r0, *d_t;
    cudaErrorCheck(hipMalloc((void **)&d_col, nz*sizeof(int)));
    cudaErrorCheck(hipMalloc((void **)&d_row, (N+1)*sizeof(int)));
    cudaErrorCheck(hipMalloc((void **)&d_val, nz*sizeof(float)));
    cudaErrorCheck(hipMalloc((void **)&d_x, N*sizeof(float)));
    cudaErrorCheck(hipMalloc((void **)&d_p, N*sizeof(float)));
    cudaErrorCheck(hipMalloc((void **)&d_q, N*sizeof(float)));
    cudaErrorCheck(hipMalloc((void **)&d_r, N*sizeof(float)));
    cudaErrorCheck(hipMalloc((void **)&d_r0, N*sizeof(float)));
    cudaErrorCheck(hipMalloc((void **)&d_t, N*sizeof(float)));

    // Get data into working device arrays, assuming x0 = 0.0
    hipMemcpy(d_row, I, (N+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col, J, nz*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_val, val, nz*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_r, b, N*sizeof(float), hipMemcpyHostToDevice);

    const float zero = 0.0;
    const float one = 1.0;
#if __CUDACC_VER_MAJOR__ >= 11
    hipsparseSpMatDescr_t descr_A;
    hipsparseDnVecDescr_t descr_p, descr_q, descr_r, descr_t;
    void *spmvBuffer = NULL;
    size_t spmvBufferSize = 0;
    hipsparseCreateCsr(&descr_A, N, N, nz, d_row, d_col, d_val,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    hipsparseCreateDnVec(&descr_p, N, d_p, HIP_R_32F);
    hipsparseCreateDnVec(&descr_q, N, d_q, HIP_R_32F);
    hipsparseCreateDnVec(&descr_r, N, d_r, HIP_R_32F);
    hipsparseCreateDnVec(&descr_t, N, d_t, HIP_R_32F);    
    hipsparseSpMV_bufferSize(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            &one, descr_A, descr_p, &zero, descr_q,
                            HIP_R_32F, 
#if __CUDACC_VER_MAJOR__ == 11 && __CUDACC_VER_MINOR__ <= 2
                            HIPSPARSE_MV_ALG_DEFAULT,
#else
                            HIPSPARSE_SPMV_ALG_DEFAULT,
#endif
                            &spmvBufferSize);
    hipMalloc(&spmvBuffer, spmvBufferSize);
#else        
    // Description of the A matrix as in Ax = b
    hipsparseMatDescr_t descrA = 0;
    cusparseStatus = hipsparseCreateMatDescr(&descrA);
    cusparseErrorCheck(cusparseStatus);
    // Define the properties of the A matrix
    hipsparseSetMatType(descrA,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA,HIPSPARSE_INDEX_BASE_ZERO);
#endif
    
    // CG iteration
    // initialize norm, p, r
    float nrmr0, nrmr;
    hipblasScopy(cublasHandle, N, d_r, 1, d_r0, 1);
    hipblasScopy(cublasHandle, N, d_r, 1, d_p, 1);
    hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &nrmr0);
    nrmr = nrmr0;
    float rhop, rho = nrmr0;
    int k;
    for (k = 0; k < maxiter; k++) {
        float alpha, beta, omega;
        float temp1, temp2;

        // compute q=Ap & alpha
#if __CUDACC_VER_MAJOR__ >= 11
        hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                     &one, descr_A, descr_p, &zero, descr_q,
                     HIP_R_32F, 
#if __CUDACC_VER_MAJOR__ == 11 && __CUDACC_VER_MINOR__ <= 2
                     HIPSPARSE_MV_ALG_DEFAULT, 
#else
                     HIPSPARSE_SPMV_ALG_DEFAULT,                     
#endif
                     spmvBuffer);
#else                
	hipsparseScsrmv(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE,N,N,nz,&one,descrA,d_val,d_row,d_col,d_p,&zero,d_q);
#endif        
	hipblasSdot(cublasHandle, N, d_r0, 1, d_q, 1, &temp1);
	alpha = rho/temp1;
        // compute s = r - \alpha q, note s is just r
        float nalpha = -alpha;
        hipblasSaxpy(cublasHandle, N, &nalpha, d_q, 1, d_r, 1);
        // compute omega = (t^{T} s) / (t^{T} t), t = As
#if __CUDACC_VER_MAJOR__ >= 11
        hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                     &one, descr_A, descr_r, &zero, descr_t,
                     HIP_R_32F, 
#if __CUDACC_VER_MAJOR__ == 11 && __CUDACC_VER_MINOR__ <= 2
                     HIPSPARSE_MV_ALG_DEFAULT, 
#else
                     HIPSPARSE_SPMV_ALG_DEFAULT,
#endif
                     spmvBuffer);
#else                
        hipsparseScsrmv(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE,N,N,nz,&one,descrA,d_val,d_row,d_col,d_r,&zero,d_t);
#endif        
        hipblasSdot(cublasHandle, N, d_t, 1, d_r, 1, &temp1);
        hipblasSdot(cublasHandle, N, d_t, 1, d_t, 1, &temp2);
        omega = temp1/temp2;
        float nomega = -omega;
        // update x = x + alpha p + omega s
        // update r = s - omega t, t = As
	hipblasSaxpy(cublasHandle, N, &alpha, d_p, 1, d_x, 1);
	hipblasSaxpy(cublasHandle, N, &omega, d_r, 1, d_x, 1);
        hipblasSaxpy(cublasHandle, N, &nomega, d_t, 1, d_r, 1);
        // compute new norm
        hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &nrmr);
        //printf("itn %d residue %e rel res %e accept %e\n", k, nrmr, sqrt(nrmr/nrmr0), sqrt(tol2));
        if (nrmr < tol2*nrmr0){
            break;
        }
        // compute beta = (rho_i/rho_i-1) (alpha/omega)
        rhop = rho;
        hipblasSdot(cublasHandle, N, d_r0, 1, d_r, 1, &rho);
        beta = (rho/rhop)*(alpha/omega);
        // compute p = r + beta (p - omega q), q = Ap
        hipblasSaxpy(cublasHandle, N, &nomega, d_q, 1, d_p, 1);
        hipblasSscal(cublasHandle, N, &beta, d_p, 1);
        hipblasSaxpy(cublasHandle, N, &one, d_r, 1, d_p, 1);
    }

    hipMemcpy(x, d_x, N*sizeof(float), hipMemcpyDeviceToHost);
    *itn = k;
    *residual = sqrt(nrmr); // sqrt it to be consistent with 1 norm in pbsa

#if __CUDACC_VER_MAJOR__ >= 11
    hipsparseDestroySpMat(descr_A);
    hipsparseDestroyDnVec(descr_p);
    hipsparseDestroyDnVec(descr_q);
    hipsparseDestroyDnVec(descr_r);
    hipsparseDestroyDnVec(descr_t);    
    hipFree(spmvBuffer);
#endif    
    
    // Destroy contexts
    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);

    // Free device memory
    hipFree(d_col);
    hipFree(d_row);
    hipFree(d_val);
    hipFree(d_x);
    hipFree(d_p);
    hipFree(d_q);
    hipFree(d_r);
    hipFree(d_r0);
    hipFree(d_t);

    // clean up all state, flush all profile data
    hipDeviceReset();

}
