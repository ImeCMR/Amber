#include "hip/hip_runtime.h"
#include "copyright.i"

//---------------------------------------------------------------------------------------------
// AMBER NVIDIA CUDA GPU IMPLEMENTATION: PMEMD VERSION
//
// July 2017, by Scott Le Grand, David S. Cerutti, Daniel J. Mermelstein, Charles Lin, and
//               Ross C. Walker
//---------------------------------------------------------------------------------------------
#ifndef AMBER_PLATFORM_AMD
#include <hip/hip_runtime.h>
#endif
#include "gpu.h"

// Use global instance instead of a local copy
#include "simulationConst.h"
CSIM_STO simulationConst cSim;

//---------------------------------------------------------------------------------------------
// Atom: structure to store SHAKE-critical atom position and mass information
//---------------------------------------------------------------------------------------------
struct Atom
{
  double invMassI;
  double xpl;
  double ypl;
  double zpl;
  double xil;
  double yil;
  double zil;
};

#if !defined(__HIPCC_RDC__)

//---------------------------------------------------------------------------------------------
// SetkShakeSim: upload critical SHAKE data to the GPU
//---------------------------------------------------------------------------------------------
void SetkShakeSim(gpuContext gpu)
{
  hipError_t status;
  status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(simulationConst));
  RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

//---------------------------------------------------------------------------------------------
// GetkShakeSim: download critical SHAKE data from the GPU
//
// Arguments:
//   gpu: overarching type for storing all parameters, coordinates, and the energy function
//
// This appears to be a debugging function.
//---------------------------------------------------------------------------------------------
void GetkShakeSim(gpuContext gpu)
{
  hipError_t status;
  status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(simulationConst));
  RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

#endif

//---------------------------------------------------------------------------------------------
// Kernels for general, PME-specialized, and hardware-dependent flavors of SHAKE, with and
// without hydrogen mass repartitioning (HMR).
//---------------------------------------------------------------------------------------------
__global__ void
__LAUNCH_BOUNDS__(SHAKE_THREADS_PER_BLOCK, SHAKE_BLOCKS)
kShake_kernel()
#include "kShake.h"

#define TISHAKE2
__global__ void
__launch_bounds__(SHAKE_THREADS_PER_BLOCK, SHAKE_BLOCKS)
kTIShake2_kernel()
#include "kShake.h"
#undef TISHAKE2

//---------------------------------------------------------------------------------------------
#define SHAKE_NEIGHBORLIST
#define TISHAKE2
__global__ void
__launch_bounds__(SHAKE_THREADS_PER_BLOCK, SHAKE_BLOCKS)
kPMETIShake2_kernel()
#include "kShake.h"
#undef TISHAKE2
__global__ void
__LAUNCH_BOUNDS__(SHAKE_THREADS_PER_BLOCK, SHAKE_BLOCKS)
kPMEShake_kernel()
#include "kShake.h"
#undef SHAKE_NEIGHBORLIST

//---------------------------------------------------------------------------------------------
#define NODPTEXTURE
#define TISHAKE2
__global__ void
__launch_bounds__(SHAKE_THREADS_PER_BLOCK, SHAKE_BLOCKS)
kTIShake2NoDPTexture_kernel()
#include "kShake.h"
#undef TISHAKE2
__global__ void
__LAUNCH_BOUNDS__(SHAKE_THREADS_PER_BLOCK, SHAKE_BLOCKS)
kShakeNoDPTexture_kernel()
#include "kShake.h"

//---------------------------------------------------------------------------------------------
#define SHAKE_NEIGHBORLIST
#define TISHAKE2
__global__ void
__launch_bounds__(SHAKE_THREADS_PER_BLOCK, SHAKE_BLOCKS)
kPMETIShake2NoDPTexture_kernel()
#include "kShake.h"
#undef TISHAKE2
__global__ void
__LAUNCH_BOUNDS__(SHAKE_THREADS_PER_BLOCK, SHAKE_BLOCKS)
kPMEShakeNoDPTexture_kernel()
#include "kShake.h"
#undef SHAKE_NEIGHBORLIST
#undef NODPTEXTURE

//---------------------------------------------------------------------------------------------
#define SHAKE_HMR
#define TISHAKE2
__global__ void
__launch_bounds__(SHAKE_THREADS_PER_BLOCK, SHAKE_BLOCKS)
kTIShake2HMR_kernel()
#include "kShake.h"
#undef TISHAKE2
__global__ void
__LAUNCH_BOUNDS__(SHAKE_THREADS_PER_BLOCK, SHAKE_BLOCKS)
kShakeHMR_kernel()
#include "kShake.h"

//---------------------------------------------------------------------------------------------
#define SHAKE_NEIGHBORLIST
#define TISHAKE2
__global__ void
__launch_bounds__(SHAKE_THREADS_PER_BLOCK, SHAKE_BLOCKS)
kPMETIShake2HMR_kernel()
#include "kShake.h"
#undef TISHAKE2
__global__ void
__LAUNCH_BOUNDS__(SHAKE_THREADS_PER_BLOCK, SHAKE_BLOCKS)
kPMEShakeHMR_kernel()
#include "kShake.h"
#undef SHAKE_NEIGHBORLIST

//---------------------------------------------------------------------------------------------
#define NODPTEXTURE
#define TISHAKE2
__global__ void
__launch_bounds__(SHAKE_THREADS_PER_BLOCK, SHAKE_BLOCKS)
kTIShake2HMRNoDPTexture_kernel()
#include "kShake.h"
#undef TISHAKE2
__global__ void
__LAUNCH_BOUNDS__(SHAKE_THREADS_PER_BLOCK, SHAKE_BLOCKS)
kShakeHMRNoDPTexture_kernel()
#include "kShake.h"

//---------------------------------------------------------------------------------------------
#define SHAKE_NEIGHBORLIST
#define TISHAKE2
__global__ void
__launch_bounds__(SHAKE_THREADS_PER_BLOCK, SHAKE_BLOCKS)
kPMETIShake2HMRNoDPTexture_kernel()
#include "kShake.h"
#undef TISHAKE2
__global__ void
__LAUNCH_BOUNDS__(SHAKE_THREADS_PER_BLOCK, SHAKE_BLOCKS)
kPMEShakeHMRNoDPTexture_kernel()
#include "kShake.h"
#undef SHAKE_NEIGHBORLIST
#undef NODPTEXTURE
#undef SHAKE_HMR

__global__ void
__launch_bounds__(UPDATE_THREADS_PER_BLOCK, 1)
kShakeOldPos_kernel()
#include "kOldShakePos.h"

//---------------------------------------------------------------------------------------------
#define SHAKE_NEIGHBORLIST
__global__ void
__launch_bounds__(UPDATE_THREADS_PER_BLOCK, 1)
kPMEShakeOldPos_kernel()
#include "kOldShakePos.h"
#undef SHAKE_NEIGHBORLIST

//---------------------------------------------------------------------------------------------
// Kernels for general, PME-specialized, and hardware-dependent flavors of RATTLE, with and
// without hydrogen mass repartitioning (HMR).
//---------------------------------------------------------------------------------------------
__global__ void
__launch_bounds__(SHAKE_THREADS_PER_BLOCK, SHAKE_BLOCKS)
kRattle_kernel(PMEFloat dt)
#include "kRattle.h"

//---------------------------------------------------------------------------------------------
#define RATTLE_NEIGHBORLIST
__global__ void
__launch_bounds__(SHAKE_THREADS_PER_BLOCK, SHAKE_BLOCKS)
kPMERattle_kernel(PMEFloat dt)
#include "kRattle.h"
#undef RATTLE_NEIGHBORLIST

//---------------------------------------------------------------------------------------------
//#define NODPTEXTURE
//__global__ void
//__launch_bounds__(SHAKE_THREADS_PER_BLOCK, SHAKE_BLOCKS)
//kShakeNoDPTexture_kernel()
//#include "kShake.h"

//---------------------------------------------------------------------------------------------
//#define SHAKE_NEIGHBORLIST
//__global__ void
//__launch_bounds__(SHAKE_THREADS_PER_BLOCK, SHAKE_BLOCKS)
//kPMEShakeNoDPTexture_kernel()
//#include "kShake.h"
//#undef SHAKE_NEIGHBORLIST
//#undef NODPTEXTURE

//---------------------------------------------------------------------------------------------
#define RATTLE_HMR
__global__ void
__launch_bounds__(SHAKE_THREADS_PER_BLOCK, SHAKE_BLOCKS)
kRattleHMR_kernel(PMEFloat dt)
#include "kRattle.h"

//---------------------------------------------------------------------------------------------
#define RATTLE_NEIGHBORLIST
__global__ void
__launch_bounds__(SHAKE_THREADS_PER_BLOCK, SHAKE_BLOCKS)
kPMERattleHMR_kernel(PMEFloat dt)
#include "kRattle.h"
#undef RATTLE_NEIGHBORLIST

//---------------------------------------------------------------------------------------------
//#define NODPTEXTURE
//__global__ void
//__launch_bounds__(SHAKE_THREADS_PER_BLOCK, SHAKE_BLOCKS)
//kShakeHMRNoDPTexture_kernel()
//#include "kShake.h"

//---------------------------------------------------------------------------------------------
//#define SHAKE_NEIGHBORLIST
//__global__ void
//__launch_bounds__(SHAKE_THREADS_PER_BLOCK, SHAKE_BLOCKS)
//kPMEShakeHMRNoDPTexture_kernel()
//#include "kShake.h"
//#undef SHAKE_NEIGHBORLIST
//#undef NODPTEXTURE
#undef RATTLE_HMR

//---------------------------------------------------------------------------------------------
// kShakeInitKernels: initialize SHAKE-reltaed kernels
//
// Arguments:
//   gpu: overarching type for storing all parameters, coordinates, and the energy function
//---------------------------------------------------------------------------------------------
void kShakeInitKernels(gpuContext gpu)
{
  if (gpu->sm_version >= SM_3X) {
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kShake_kernel), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kPMEShake_kernel), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kShakeNoDPTexture_kernel), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kPMEShakeNoDPTexture_kernel), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kShakeHMR_kernel), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kPMEShakeHMR_kernel), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kShakeHMRNoDPTexture_kernel), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kPMEShakeHMRNoDPTexture_kernel), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kTIShake2_kernel), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kPMETIShake2_kernel), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kTIShake2NoDPTexture_kernel), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kPMETIShake2NoDPTexture_kernel), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kTIShake2HMR_kernel), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kPMETIShake2HMR_kernel), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kTIShake2HMRNoDPTexture_kernel), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kPMETIShake2HMRNoDPTexture_kernel), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kRattle_kernel), hipFuncCachePreferL1); //added by zhf, used for middle-scheme
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kPMERattle_kernel), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kRattleHMR_kernel), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kPMERattleHMR_kernel), hipFuncCachePreferL1);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kShake_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kPMEShake_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kShakeNoDPTexture_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kPMEShakeNoDPTexture_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kShakeHMR_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kPMEShakeHMR_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kShakeHMRNoDPTexture_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kPMEShakeHMRNoDPTexture_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kTIShake2_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kPMETIShake2_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kTIShake2NoDPTexture_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kPMETIShake2NoDPTexture_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kTIShake2HMR_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kPMETIShake2HMR_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kTIShake2HMRNoDPTexture_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kPMETIShake2HMRNoDPTexture_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kRattle_kernel), hipSharedMemBankSizeEightByte); //added by zhf, used for middle-scheme
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kPMERattle_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kRattleHMR_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kPMERattleHMR_kernel), hipSharedMemBankSizeEightByte);
  }
}

//---------------------------------------------------------------------------------------------
// kShake: implement the various SHAKE kernels enumerated above.
//
// Arguments:
//   gpu: overarching type for storing all parameters, coordinates, and the energy function
//---------------------------------------------------------------------------------------------
void kShake(gpuContext gpu)
{
  unsigned int totalBlocks = (gpu->sim.slowShakeOffset + gpu->shakeThreadsPerBlock - 1) /
                             gpu->shakeThreadsPerBlock;
  if(gpu->sim.tishake == 2) {
    totalBlocks = (gpu->sim.slowTIShakeOffset + gpu->shakeThreadsPerBlock - 1) /
                             gpu->shakeThreadsPerBlock;
  }
  int launchBlocks = 65535;

  if (gpu->bNeighborList) {
      kPMEShakeOldPos_kernel<<<gpu->updateBlocks, gpu->updateThreadsPerBlock>>>();
  }
  else {
      kShakeOldPos_kernel<<<gpu->updateBlocks, gpu->updateThreadsPerBlock>>>();
  }


  if (gpu->bNoDPTexture) {
    while (totalBlocks > 0) {
      int blocks = min(totalBlocks, launchBlocks);
      if (gpu->bUseHMR) {
        if (gpu->bNeighborList) {
          if (gpu->sim.tishake == 2) {
            kPMETIShake2HMRNoDPTexture_kernel<<<blocks, gpu->shakeThreadsPerBlock>>>();
          }
          else {
            kPMEShakeHMRNoDPTexture_kernel<<<blocks, gpu->shakeThreadsPerBlock>>>();
          }
        }
        else {
          if (gpu->sim.tishake == 2) {
            kTIShake2HMRNoDPTexture_kernel<<<blocks, gpu->shakeThreadsPerBlock>>>();
          }
          else {
            kShakeHMRNoDPTexture_kernel<<<blocks, gpu->shakeThreadsPerBlock>>>();
          }
        }
      }
      else {
        if (gpu->bNeighborList) {
          if (gpu->sim.tishake == 2) {
            kPMETIShake2NoDPTexture_kernel<<<blocks, gpu->shakeThreadsPerBlock>>>();
          }
          else {
            kPMEShakeNoDPTexture_kernel<<<blocks, gpu->shakeThreadsPerBlock>>>();
          }
        }
        else {
          if (gpu->sim.tishake == 2) {
            kTIShake2NoDPTexture_kernel<<<blocks, gpu->shakeThreadsPerBlock>>>();
          }
          else {
            kShakeNoDPTexture_kernel<<<blocks, gpu->shakeThreadsPerBlock>>>();
          }
        }
      }
      LAUNCHERROR("kShakeNoDPTexture");
      totalBlocks -= blocks;
    }
  }
  else {
    while (totalBlocks > 0) {
      int blocks = min(totalBlocks, launchBlocks);
      if (gpu->bUseHMR) {
        if (gpu->bNeighborList) {
          if (gpu->sim.tishake == 2) {
            kPMETIShake2HMR_kernel<<<blocks, gpu->shakeThreadsPerBlock>>>();
          }
          else {
            kPMEShakeHMR_kernel<<<blocks, gpu->shakeThreadsPerBlock>>>();
          }
        }
        else {
          if (gpu->sim.tishake == 2) {
            kTIShake2HMR_kernel<<<blocks, gpu->shakeThreadsPerBlock>>>();
          }
          else {
            kShakeHMR_kernel<<<blocks, gpu->shakeThreadsPerBlock>>>();
          }
        }
      }
      else {
        if (gpu->bNeighborList) {
          if (gpu->sim.tishake == 2) {
            kPMETIShake2_kernel<<<blocks, gpu->shakeThreadsPerBlock>>>();
          }
          else {
            kPMEShake_kernel<<<blocks, gpu->shakeThreadsPerBlock>>>();
          }
        }
        else {
          if (gpu->sim.tishake == 2) {
            kTIShake2_kernel<<<blocks, gpu->shakeThreadsPerBlock>>>();
          }
          else {
            kShake_kernel<<<blocks, gpu->shakeThreadsPerBlock>>>();
          }
        }
      }
      LAUNCHERROR("kShake");
      totalBlocks -= blocks;
    }
  }
}

//---------------------------------------------------------------------------------------------
// kRattle: implement the various RATTLE kernels enumerated above.
//
// Arguments:
//   gpu: overarching type for storing all parameters, coordinates, and the energy function
//---------------------------------------------------------------------------------------------
void kRattle(gpuContext gpu, PMEFloat dt)
{
  unsigned int totalBlocks = (gpu->sim.slowShakeOffset + gpu->shakeThreadsPerBlock - 1) /
                             gpu->shakeThreadsPerBlock;
  int launchBlocks = 65535;
  //if (gpu->bNoDPTexture) {
  //  while (totalBlocks > 0) {
  //    int blocks = min(totalBlocks, launchBlocks);
  //    if (gpu->bUseHMR) {
  //      if (gpu->bNeighborList) {
  //        kPMEShakeHMRNoDPTexture_kernel<<<blocks, gpu->shakeThreadsPerBlock>>>();
  //      }
  //      else {
  //        kShakeHMRNoDPTexture_kernel<<<blocks, gpu->shakeThreadsPerBlock>>>();
  //      }
  //    }
  //    else {
  //      if (gpu->bNeighborList) {
  //        kPMEShakeNoDPTexture_kernel<<<blocks, gpu->shakeThreadsPerBlock>>>();
  //      }
  //      else {
  //        kShakeNoDPTexture_kernel<<<blocks, gpu->shakeThreadsPerBlock>>>();
  //      }
  //    }
  //    LAUNCHERROR("kShakeNoDPTexture");
  //    totalBlocks -= blocks;
  //  }
  //}
  //else {
    while (totalBlocks > 0) {
      int blocks = min(totalBlocks, launchBlocks);
      if (gpu->bUseHMR) {
        if (gpu->bNeighborList) {
          kPMERattleHMR_kernel<<<blocks, gpu->shakeThreadsPerBlock>>>(dt);
        }
        else {
          kRattleHMR_kernel<<<blocks, gpu->shakeThreadsPerBlock>>>(dt);
        }
      }
      else {
        if (gpu->bNeighborList) {
          kPMERattle_kernel<<<blocks, gpu->shakeThreadsPerBlock>>>(dt);
        }
        else {
          kRattle_kernel<<<blocks, gpu->shakeThreadsPerBlock>>>(dt);
        }
      }
      LAUNCHERROR("kRattle");
      totalBlocks -= blocks;
    }
  //}
}
